#include "hip/hip_runtime.h"
#include "main.h"

__global__ void euler(float *results)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    Tapn net;
    Place place1;
    float token = 0.0f;
    float tokens[1]{token};
    place1.addTokens(tokens, 1);
    Place place2;

    Arc arc1;
    arc1.place = &place1;
    arc1.type = TRANSPORT;
    arc1.timings[0] = 0.0f;
    arc1.timings[1] = FLT_MAX;

    OutputArc oArc1;
    oArc1.isTransport = true;
    oArc1.output = &place1;

    Distribution dis1;
    dis1.type = UNIFORM;
    dis1.a = 0.0f;
    dis1.b = 1.0f;
    dis1.init();

    Transition trans1;
    trans1.distribution = &dis1;
    trans1.inputArcs[0] = &arc1;
    trans1.inputArcsCount++;
    trans1.outputArcs[0] = &oArc1;
    trans1.outputArcsCount++;

    Distribution dis2;
    dis2.type = CONSTANT;
    dis2.a = 0.0f;

    Arc arc2;
    arc2.place = &place1;
    arc2.type = INPUT;
    arc2.timings[0] = 1.0f;
    arc2.timings[1] = FLT_MAX;

    OutputArc oArc2;
    oArc2.isTransport = false;
    oArc2.output = &place2;

    Transition trans2;
    trans2.distribution = &dis2;
    trans2.inputArcs[0] = &arc2;
    trans2.inputArcsCount++;
    trans2.outputArcs[0] = &oArc2;
    trans2.outputArcsCount++;

    Place *places[2]{&place1, &place2};

    net.places = places;

    net.placesCount = 2;

    Transition *transitions[2]{&trans1, &trans2};
    net.transitions = transitions;
    net.transitionsCount = 2;
    // TokenAgeObserver tokenAgeObs(MAXFLOAT);
    // net.addObserver(&tokenAgeObs);
    // TokenCountObserver tokenCountObs;
    // net.addObserver(&tokenCountObs);
    net.run();
    results[tid] += net.steps - 1;
    // net.step(&test);
    // //printf("\n place 0 %f\n", place1.tokens[0]);
    // net.step(&test);
    // net.step(&test);
}

__global__ void sum(float *array, int numSimulations, int totalThreads)
{
    float total = 0.0f;

    for (int i = 0; i < totalThreads; i++)
    {
        total += array[i];
    }
    printf("euler value is %.11f\n", (double)total / numSimulations);
    printf("real euler is 2.71828");
}
__global__ void summage(float *array, int numSimulations, int totalThreads)
{
    int tid = threadIdx.x;
    float sum = 0.0f;

    for (int i = 0; i < numSimulations / totalThreads; i++)
    {
        sum += array[tid + i * totalThreads];
    }

    array[tid] = sum;
}
int main(int argc, char *argv[])
{
    auto start = std::chrono::high_resolution_clock::now();
    float confidence;
    float error;
    int threads = 512;
    int blockCount = 2048;
    if (argc < 3)
    {
        confidence = 0.95f;
        error = 0.0005f;
    }
    else
    {
        confidence = std::stof(argv[1]);
        error = std::stof(argv[2]);
    }
    std::cout << "confidence: " << confidence << " error: " << error << std::endl;
    float number = ceil((log(2 / (1 - confidence))) / (2 * error * error));
    std::cout << "number of executions: " << number << std::endl;
    int loopCount = ceil(number / (blockCount * threads));
    std::cout << "loop count: " << loopCount << std::endl;
    std::cout << "number of executions: " << loopCount * blockCount * threads << std::endl;
    float *d_results;

    hipMalloc((void **)&d_results, blockCount * threads * sizeof(float));
    hipMemset((void **)&d_results, 0, blockCount * threads * sizeof(float));
    for (size_t i = 0; i < loopCount; i++)
    {
        euler<<<blockCount, threads>>>(d_results);
        hipDeviceSynchronize();
    }

    summage<<<1, threads>>>(d_results, blockCount * threads, threads);
    hipDeviceSynchronize();
    sum<<<1, 1>>>(d_results, loopCount * blockCount * threads, threads);
    hipDeviceSynchronize();
    hipError_t errSync = hipDeviceSynchronize();
    hipError_t errAsync = hipGetLastError();

    if (errSync != hipSuccess)
    {
        printf("Sync error: %s\n", hipGetErrorString(errSync));
    }
    if (errAsync != hipSuccess)
    {
        printf("Launch error: %s\n", hipGetErrorString(errAsync));
    }
    hipFree(d_results);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "time run: " << duration.count() << std::endl;
    return 0;
}