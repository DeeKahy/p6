#include "hip/hip_runtime.h"
#include "euler.h"
#define checkCudaErrors(call)                                                            \
    {                                                                                    \
        hipError_t err = call;                                                          \
        if (err != hipSuccess)                                                          \
        {                                                                                \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl;                           \
            exit(EXIT_FAILURE);                                                          \
        }                                                                                \
    }
__global__ void euler(float *results)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    Tapn net;
    Transition transitions[2];
    net.currentTime = 0.0f;
    net.steps = 0;
    Place places[2];
    float token = 0.0f;
    float tokens[1]{token};
    places[0].addTokens(tokens, 1);
    transitions[0].distribution.a = 0.0f;
    transitions[0].distribution.b = 1.0f;
    transitions[0].distribution.type = UNIFORM;

    transitions[0].inputArcs[0].place = 0;
    transitions[0].inputArcs[0].type = TRANSPORT;
    transitions[0].inputArcs[0].timings[0] = 0.0f;
    transitions[0].inputArcs[0].timings[1] = FLT_MAX;
    transitions[0].inputArcsCount++;
    transitions[0].outputArcs[0].isTransport = true;
    transitions[0].outputArcs[0].output = 0;
    transitions[0].outputArcsCount++;

    transitions[1].distribution.a = 0.0f;
    transitions[1].distribution.type = CONSTANT;
    transitions[1].inputArcs[0].place = 0;
    transitions[1].inputArcs[0].type = INPUT;
    transitions[1].inputArcs[0].timings[0] = 1.0f;
    transitions[1].inputArcs[0].timings[1] = FLT_MAX;
    transitions[1].inputArcsCount++;
    transitions[1].outputArcs[0].output = 1;
    transitions[1].outputArcsCount++;

    net.placesCount = 2;
    net.transitions = transitions;
    net.transitionsCount = 2;
    // TokenAgeObserver tokenAgeObs(MAXFLOAT);
    // net.addObserver(&tokenAgeObs);
    // TokenCountObserver tokenCountObs;
    // net.addObserver(&tokenCountObs);

    net.run(places);
    // printf("\n%f\n",net.currentTime);
    results[tid] += net.steps;

    // net.step(&test);
    // //printf("\n place 0 %f\n", place1.tokens[0]);
    // net.step(&test);
    // net.step(&test);
}

__global__ void sum(float *array, unsigned long long numSimulations, unsigned long long totalThreads)
{
    double total = 0.0f;

    for (int i = 0; i < totalThreads; i++)
    {
        total += array[i];
    }
    printf("euler value is %.11f\n", total / numSimulations);
    printf("real euler is 2.71828\n");
}
__global__ void summage(float *array, unsigned long long numSimulations, unsigned long long totalThreads)
{
    int tid = threadIdx.x;
    double sum = 0.0f;

    for (int i = 0; i < numSimulations / totalThreads; i++)
    {
        sum += array[tid + i * totalThreads];
    }

    array[tid] = sum;
}
int main(int argc, char *argv[])
{
    auto start = std::chrono::high_resolution_clock::now();
    float confidence;
    float error;
    unsigned long long threads = 1024;
    unsigned long long blockCount = 2048;
    if (argc < 3)
    {
        confidence = 0.95f;
        error = 0.005f;
    }
    else
    {
        confidence = std::stof(argv[1]);
        error = std::stof(argv[2]);
    }
    std::cout << "confidence: " << confidence << " error: " << error << "\n";
    float number = ceil((log(2 / (1 - confidence))) / (2 * error * error));
    std::cout << "execution calculated: " << number << "\n";
    unsigned long long loopCount = ceil(number / (blockCount * threads));
    std::cout << "loop count: " << loopCount << "\n";
    unsigned long long N{blockCount * threads};
    std::cout << "number of executions run: " << N * loopCount << "\n";
    float *d_results;
    hipMalloc((void **)&d_results, N * sizeof(float));
    hipMemset(d_results, 0, N * sizeof(float));
    for (size_t i = 0; i < loopCount; i++)
    {
        euler<<<blockCount, threads>>>(d_results);
        hipDeviceSynchronize();
    }

    thrust::device_ptr<float> d_ptr = thrust::device_pointer_cast(d_results);
    double tot = thrust::reduce(d_ptr, d_ptr + N);
    std::cout << "Success rate: " << tot / (N * loopCount) << "\n";
    hipError_t errSync = hipDeviceSynchronize();
    hipError_t errAsync = hipGetLastError();

    if (errSync != hipSuccess)
    {
        printf("Sync error: %s\n", hipGetErrorString(errSync));
    }
    if (errAsync != hipSuccess)
    {
        printf("Launch error: %s\n", hipGetErrorString(errAsync));
    }
    hipFree(d_results);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "time run: " << duration.count() << "\n";
    return 0;
}