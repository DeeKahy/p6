#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <chrono>
// Helper function to check CUDA errors
#define checkCudaErrors(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                 << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

struct Transition
{
	int type;
	int from;
	int to;
	float guard[2];
	void(*function)(float*, float*);
};

struct Euler
{
	int timesFired{ 0 };
	int timesCalled{ 0 };
	float places[2]{ 0 };
	bool success{ false };
	Transition transitions[2];
};

__device__ void reset(float* from, float* to) {
	*from = 0.0f;
	*to = 0.0f;
}

__device__ void increment(float* from, float* to) {
	hiprandState state;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(clock64() + tid, tid, 0, &state);
	float randomValue = hiprand_uniform(&state);
	*to = *from + randomValue;
};

__device__ void simulateThread(Euler* euler) {
	bool shouldBreak = false;
	euler->timesCalled += 1;
	while (!shouldBreak) {
		Transition youngest = euler->transitions[0];
		for (size_t i = 0; i < 2; i++)
		{
			if (euler->places[0] >= euler->transitions[i].guard[0] && youngest.guard[0] < euler->transitions[i].guard[0])
			{
				youngest = euler->transitions[i];
			}
		}
		youngest.function(&euler->places[youngest.from], &euler->places[youngest.to]);


		if (euler->places[youngest.from] == 0.0f) {
			shouldBreak = true;
		}
		else {
			euler->timesFired++;
		}
	}
}

__global__ void initThread(float* results) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Euler euler;

	Transition timeIncrease;
	timeIncrease.from = 0;
	timeIncrease.to = 0;
	timeIncrease.guard[0] = 0;
	timeIncrease.guard[1] = 1.0f;
	timeIncrease.function = &increment;
	euler.transitions[0] = timeIncrease;

	Transition timeOut;
	timeOut.from = 0;
	timeOut.to = 1;
	timeOut.guard[0] = 1.0f;
	timeOut.guard[1] = 100000.0f;
	timeOut.function = &reset;
	euler.transitions[1] = timeOut;
	simulateThread(&euler);

	results[tid] = euler.timesFired / euler.timesCalled;
}
__global__ void initCurandStates(hiprandState* states, unsigned long seed)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed + tid, tid, 0, &states[tid]);
}
__global__ void sum(float* array, int numSimulations) {
	float total = 0.0f;
	for (int i = 0; i < 1024; i++) {
		total += array[i];
	}
	printf("euler value is %f\n", total / numSimulations);
}
__global__ void summage(float* array, int gridSize) {
	int tid = threadIdx.x;
	float sum = 0.0f;

	for (int i = 0; i < gridSize/1024; i++) {
		sum += array[tid + i * 1024];
	}

	array[tid] = sum;
}

int main(int argc, char *argv[]) {
	auto start = std::chrono::high_resolution_clock::now();
	int gridSize = std::stoi(argv[1]);
	int blockSize = 1024;
	int numSimulations = gridSize * blockSize;
	
	float* results = new float[numSimulations];
	float* d_results;
	hipMalloc((void**)&d_results, numSimulations * sizeof(float));
	hiprandState* d_states;
	hipMalloc((void**)&d_states, numSimulations * sizeof(hiprandState));
	initCurandStates << <gridSize, blockSize >> > (d_states, time(0));
	hipDeviceSynchronize();
	initThread << <gridSize, blockSize >> > (d_results);
	hipDeviceSynchronize();

	summage << <1, blockSize >> > (d_results, numSimulations);
	hipDeviceSynchronize();
	sum << <1, 1 >> > (d_results, numSimulations);
	hipDeviceSynchronize();
	std::cout << "True value of e: 2.71828..." << std::endl;
	delete[] results;
	hipFree(d_results);
	hipFree(d_states);
	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
	return duration.count();
}