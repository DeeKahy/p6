#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>

// Helper function to check CUDA errors
#define checkCudaErrors(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                 << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

enum FunctionType
{
    TIME_INCREASE,
    TIME_OUT
};

struct Transition
{
    int type;
    int from;
    int to;
    float guard[2];
    FunctionType functionType;
};

struct Euler
{
    int places[2]{ 0 };
    Transition transitions[2];
};

__device__ void timeIncreaseFunction(float* value, hiprandState* state)
{
    float randomValue = hiprand_uniform(state);
    *value += randomValue;
}

__device__ void timeOutFunction(float* value, hiprandState* state)
{
    // No operation needed for timeOut
}

__device__ void callFunction(FunctionType functionType, float* value, hiprandState* state)
{
    switch (functionType)
    {
    case TIME_INCREASE:
        timeIncreaseFunction(value, state);
        break;
    case TIME_OUT:
        timeOutFunction(value, state);
        break;
    }
}

__global__ void simulate(Euler* euler, int* counts, float* values, hiprandState* states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState* state = &states[tid];

    float test = 0.0f;
    int timesFired = 0;
    bool shouldBreak = false;

    while (!shouldBreak) {
        // Always select the TIME_INCREASE transition until test > 1.0f
        Transition youngest = euler->transitions[0];
        for (size_t i = 0; i < 2; i++)
        {
            if (test >= euler->transitions[i].guard[0] && youngest.guard[0] < euler->transitions[i].guard[0])
            {
                youngest = euler->transitions[i];
            }
        }
        callFunction(youngest.functionType, &test, state);
        timesFired++;
        if (test >= 1.0f) {
            shouldBreak = true;
        }
    }

    // Store the count (this approximates e)
    counts[tid] += 1;
    values[tid] += timesFired;
}

__global__ void initCurandStates(hiprandState* states, unsigned long seed)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed + tid, tid, 0, &states[tid]);
}

int main()
{
    // Create Euler struct with transitions
    Euler euler;

    Transition timeIncrease;
    timeIncrease.from = 0;
    timeIncrease.to = 0;
    timeIncrease.guard[0] = 0;
    timeIncrease.guard[1] = 1.0f;
    timeIncrease.functionType = TIME_INCREASE;
    euler.transitions[0] = timeIncrease;

    Transition timeOut;
    timeOut.from = 0;
    timeOut.to = 1;
    timeOut.guard[0] = 1.0f;
    timeOut.guard[1] = std::numeric_limits<float>::max();
    timeOut.functionType = TIME_OUT;
    euler.transitions[1] = timeOut;

    // Allocate memory on the device for Euler struct
    Euler* d_euler;
    checkCudaErrors(hipMalloc((void**)&d_euler, sizeof(Euler)));
    checkCudaErrors(hipMemcpy(d_euler, &euler, sizeof(Euler), hipMemcpyHostToDevice));

    // Number of threads and blocks
    const int numThreads = 1024;
    const int numBlocks = 3000;
    const int numSimulations = numThreads * numBlocks;

    // Allocate arrays for return values
    int* d_counts;
    float* d_values;
    checkCudaErrors(hipMalloc((void**)&d_counts, numSimulations * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_values, numSimulations * sizeof(float)));

    // Initialize arrays to zero
    checkCudaErrors(hipMemset(d_counts, 0, numSimulations * sizeof(int)));
    checkCudaErrors(hipMemset(d_values, 0, numSimulations * sizeof(float)));

    // Allocate and initialize hiprand states
    hiprandState* d_states;
    checkCudaErrors(hipMalloc((void**)&d_states, numSimulations * sizeof(hiprandState)));

    std::cout << "Initializing CURAND states..." << std::endl;
    initCurandStates << <numBlocks, numThreads >> > (d_states, time(0));
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "Running simulation..." << std::endl;
    // Launch the kernel with Euler struct

    // for (size_t i = 0; i < 10000; i++)
    // {
        simulate << <numBlocks, numThreads >> > (d_euler, d_counts, d_values, d_states);
        checkCudaErrors(hipDeviceSynchronize());

    // }


    // Check for errors after kernel execution
    checkCudaErrors(hipGetLastError());

    std::cout << "Copying results back to host..." << std::endl;
    // Allocate arrays on the host
    std::vector<int> h_counts(numSimulations);
    std::vector<float> h_values(numSimulations);

    // Copy data from device to host
    checkCudaErrors(hipMemcpy(h_counts.data(), d_counts, numSimulations * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_values.data(), d_values, numSimulations * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    hipFree(d_euler);
    hipFree(d_counts);
    hipFree(d_values);
    hipFree(d_states);

    // Process results on the host
    double totalCounts = 0;
    float totalValues = 0;
    for (int i = 0; i < numSimulations; ++i)
    {
        // std::cout << h_counts[i] << " " << h_values[i] << "\n";
        totalCounts += h_counts[i];
        totalValues += h_values[i];
    }

    if (totalCounts > 0)
    {
        float approxE = totalValues / totalCounts;
        std::cout << "Approximation of e: " << approxE << std::endl;
        std::cout << "True value of e: 2.71828..." << std::endl;
        std::cout << numSimulations << std::endl;
    }
    else
    {
        std::cout << "No valid results found." << std::endl;
    }

    return 0;
}